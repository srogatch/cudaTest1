#include "hip/hip_runtime.h"
#include "cuda_interop.h"
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <>

__global__ void TestKernel() {
    printf("Hello CUDA!\n");
}

void TestOnCuda() {
    hipError_t err = hipSetDevice(0);
    if(err != 0) {
        fprintf(stderr, "hipSetDevice() returned %d: %s\n", err, hipGetErrorString(err));
    }
    TestKernel<<<1,1>>>();
    err = hipGetLastError();
    if(err != 0) {
        fprintf(stderr, "hipGetLastError() returned %d: %s\n", err, hipGetErrorString(err));
    }
    err = hipDeviceSynchronize();
    if(err != 0) {
        fprintf(stderr, "hipDeviceSynchronize() returned %d: %s\n", err, hipGetErrorString(err));
    }
}